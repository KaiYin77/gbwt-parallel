#include "hip/hip_runtime.h"
#include <chrono>
#include <thrust/device_free.h>
#include <thrust/device_malloc.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust_sort.cuh>

void print_vec(const thrust::host_vector<size_type> &vec) {
  for (auto &item : vec) {
    std::cout << item << " ";
  }
  std::cout << "\n";
}

__device__ uint64_t get_int(const uint64_t *m_data, const size_type idx,
                            const uint8_t len) {
  const uint64_t *word = m_data + (idx >> 6);
  const uint8_t offset = idx & 0x3F;
  uint64_t w1 = (*word) >> offset;
  if ((offset + len) > 64) { // if offset+len > 64
                             // w1 or w2 adepted:
    return w1 | ((*(word + 1) &
                  (1 << ((offset + len) & 0x3F)) - 1) // set higher bits zero
                 << (64 - offset));                   // move bits to the left
  } else {
    return w1 & ((1 << len) - 1);
  }
}

__global__ void test_get_int(const uint64_t *d_source, node_type *d_test,
                             const size_type vec_size,
                             const size_type start_idx, const int t_width) {
  size_type idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < vec_size) {
    d_test[idx] =
        get_int(d_source, size_type((start_idx + idx)) * t_width, t_width);
  }
  return;
}

__global__ void assign_key(const uint64_t *d_source,
                           const size_type *d_start_pos, node_type *d_keys,
                           size_type *d_seq_id, const size_type position,
                           const size_type seqs_size, const int t_width) {
  size_type idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < seqs_size) {
    d_keys[idx] = get_int(
        d_source, (d_start_pos[d_seq_id[idx]] + position) * t_width, t_width);
  }
  return;
}

struct is_zero {
  __host__ __device__ bool operator()(int x) { return (x == 0); }
};

std::vector<std::vector<std::pair<size_type, node_type>>>
radix_sort(const text_type &source, std::vector<size_type> &sequence_id,
           const std::unique_ptr<std::unordered_map<size_type, size_type>>
               &start_pos_map,
           const std::uint64_t total_nodes) {
  double init_time = 0, key_time = 0, /*h2d_copy_time = 0,*/ sort_time = 0,
         d2h_copy_time = 0, remove_time = 0, place_time = 0;
  std::chrono::steady_clock::time_point begin, end;
  begin = std::chrono::steady_clock::now();

  // lambda function to get the nearest number of power of 2
  /*
  auto round_up_32 = [](unsigned int v) {
    --v;
    v |= v >> 1;
    v |= v >> 2;
    v |= v >> 4;
    v |= v >> 8;
    v |= v >> 16;
    ++v;
    return v;
  };
  */

  std::vector<std::vector<std::pair<size_type, node_type>>> sorted_seqs(
      total_nodes);

  // ---- Prepare values to be used ---- //
  // width of the integers which are accessed via the [] operator
  const uint8_t t_width = source.width();
  const size_type source_size_byte = source.bit_size() / 8 + 1;
  const size_type seqs_size = (*start_pos_map).size(); // total sequence size

  // copy source to device memory
  // hipHostMalloc((void **)&source, source_size_byte, hipHostMallocDefault);
  uint64_t *d_source;
  hipMalloc(&d_source, source_size_byte);
  hipMemcpyAsync(d_source, source.data(), source_size_byte,
                  hipMemcpyHostToDevice);
  // hipMemcpy(d_source, source.data(), source_size_byte,
  // hipMemcpyHostToDevice);

  // copy start_position
  thrust::host_vector<size_type> start_pos;
  start_pos.reserve(seqs_size);
  for (auto &seq_id : sequence_id) {
    start_pos.push_back((*start_pos_map)[seq_id]);
  }
  thrust::device_vector<size_type> start_pos_vec = start_pos;
  size_type *d_start_pos = thrust::raw_pointer_cast(&start_pos_vec[0]);
  // thrust::device_ptr<size_type> d_start_pos =
  // start_pos_vec.data();

  // copy sequence_id
  thrust::host_vector<size_type> h_seq_id(sequence_id);
  thrust::device_vector<size_type> seq_id_vec = h_seq_id;
  thrust::device_ptr<size_type> d_seq_id =
      thrust::device_pointer_cast(&seq_id_vec[0]);
  size_type *d_seq_id_raw = thrust::raw_pointer_cast(&seq_id_vec[0]);
  // thrust::device_ptr<size_type> d_seq_id = seq_id_vec.data();

  // use gpu to assign keys
  // there about 2000~ sequences passed in
  thrust::host_vector<node_type> h_keys_vec(seqs_size);
  thrust::device_ptr<node_type> d_keys =
      thrust::device_malloc(sizeof(node_type) * seqs_size);
  node_type *d_keys_raw = thrust::raw_pointer_cast(d_keys);

  size_type arr_start_idx = 0;     // first index which is not an ENDMARKER
  size_type seqs_left = seqs_size; // sequences that have not
                                   // reached the ENDMARKER
  const int thread_per_block = 512;
  hipDeviceSynchronize();
  end = std::chrono::steady_clock::now();
  init_time = std::chrono::duration_cast<std::chrono::microseconds>(end - begin)
                  .count();
  for (size_type position = 0; seqs_left > 0; ++position) {
    //---- Assign Keys ----//
    begin = std::chrono::steady_clock::now();
    const int block_per_grid =
        (seqs_left + thread_per_block - 1) / thread_per_block;
    assign_key<<<block_per_grid, thread_per_block>>>(
        d_source, d_start_pos, d_keys_raw + arr_start_idx,
        d_seq_id_raw + arr_start_idx, position, seqs_left, t_width);
    end = std::chrono::steady_clock::now();
    key_time +=
        std::chrono::duration_cast<std::chrono::microseconds>(end - begin)
            .count();

    //---- Radix Sort ----//
    begin = std::chrono::steady_clock::now();
    thrust::stable_sort_by_key(d_keys + arr_start_idx, d_keys + seqs_size,
                               d_seq_id + arr_start_idx);
    end = std::chrono::steady_clock::now();
    sort_time +=
        std::chrono::duration_cast<std::chrono::microseconds>(end - begin)
            .count();

    //---- Copy keys and sequence id back to host ----//
    begin = std::chrono::steady_clock::now();
    thrust::copy(d_seq_id + arr_start_idx, d_seq_id + seqs_size,
                 h_seq_id.begin());
    thrust::copy(d_keys + arr_start_idx, d_keys + seqs_size,
                 h_keys_vec.begin());
    end = std::chrono::steady_clock::now();
    d2h_copy_time +=
        std::chrono::duration_cast<std::chrono::microseconds>(end - begin)
            .count();

    //---- Remove paths that reaches the ENDMARKER(zero):  version2 ----//
    // remove: find the first index in h_keys_vec that is not an ENDMARKER
    ///*
    begin = std::chrono::steady_clock::now();
    size_type end_counter = 0;
    for (size_type i = 0; i < seqs_left; ++i) {
      if (h_keys_vec[i] != gbwt::ENDMARKER) {
        break;
      } else {
        ++end_counter;
      }
    }
    arr_start_idx += end_counter;
    seqs_left = seqs_size - arr_start_idx;
    end = std::chrono::steady_clock::now();
    remove_time +=
        std::chrono::duration_cast<std::chrono::microseconds>(end - begin)
            .count();
    if (seqs_left <= 0)
      break;
    //*/
    begin = std::chrono::steady_clock::now();
    for (size_type i = end_counter; i < seqs_left; ++i) {
      size_type seq_id = h_seq_id[i];
      node_type next_node_id = source[(*start_pos_map)[seq_id] + position + 1];
      sorted_seqs[h_keys_vec[i] - 1].push_back({seq_id, next_node_id});
    }
    // const int unroll_num = 8;
    /*
    for (size_type i = end_counter; i < (seqs_left / unroll_num);
         i += unroll_num) {
      size_type seq_id = h_seq_id[i];
      node_type next_node_id = source[(*start_pos_map)[seq_id] + position + 1];
      sorted_seqs[h_keys_vec[i] - 1].push_back({seq_id, next_node_id});

      size_type seq_id1 = h_seq_id[i + 1];
      node_type next_node_id1 =
          source[(*start_pos_map)[seq_id1] + position + 1];
      sorted_seqs[h_keys_vec[i + 1] - 1].push_back({seq_id1, next_node_id1});

      size_type seq_id2 = h_seq_id[i + 2];
      node_type next_node_id2 =
          source[(*start_pos_map)[seq_id2] + position + 1];
      sorted_seqs[h_keys_vec[i + 2] - 1].push_back({seq_id2, next_node_id2});

      size_type seq_id3 = h_seq_id[i + 3];
      node_type next_node_id3 =
          source[(*start_pos_map)[seq_id3] + position + 1];
      sorted_seqs[h_keys_vec[i + 3] - 1].push_back({seq_id3, next_node_id3});

      size_type seq_id4 = h_seq_id[i + 4];
      node_type next_node_id4 =
          source[(*start_pos_map)[seq_id4] + position + 1];
      sorted_seqs[h_keys_vec[i + 4] - 1].push_back({seq_id4, next_node_id4});

      size_type seq_id5 = h_seq_id[i + 5];
      node_type next_node_id5 =
          source[(*start_pos_map)[seq_id5] + position + 1];
      sorted_seqs[h_keys_vec[i + 5] - 1].push_back({seq_id5, next_node_id5});

      size_type seq_id6 = h_seq_id[i + 6];
      node_type next_node_id6 =
          source[(*start_pos_map)[seq_id6] + position + 1];
      sorted_seqs[h_keys_vec[i + 6] - 1].push_back({seq_id6, next_node_id6});

      size_type seq_id7 = h_seq_id[i + 7];
      node_type next_node_id7 =
          source[(*start_pos_map)[seq_id7] + position + 1];
      sorted_seqs[h_keys_vec[i + 7] - 1].push_back({seq_id7, next_node_id7});

      size_type seq_id1 = h_seq_id[i + 1];
      node_type next_node_id1 = source[(*start_pos_map)[seq_id1] + position +
      1];
      sorted_seqs[h_keys_vec[i + 1] - 1].push_back({seq_id1, next_node_id1});

      size_type seq_id1 = h_seq_id[i + 1];
      node_type next_node_id1 = source[(*start_pos_map)[seq_id1] + position +
      1];
      sorted_seqs[h_keys_vec[i + 1] - 1].push_back({seq_id1, next_node_id1});

      size_type seq_id1 = h_seq_id[i + 1];
      node_type next_node_id1 = source[(*start_pos_map)[seq_id1] + position +
      1];
      sorted_seqs[h_keys_vec[i + 1] - 1].push_back({seq_id1, next_node_id1});

      size_type seq_id1 = h_seq_id[i + 1];
      node_type next_node_id1 = source[(*start_pos_map)[seq_id1] + position +
      1];
      sorted_seqs[h_keys_vec[i + 1] - 1].push_back({seq_id1, next_node_id1});

      size_type seq_id1 = h_seq_id[i + 1];
      node_type next_node_id1 = source[(*start_pos_map)[seq_id1] + position +
      1];
      sorted_seqs[h_keys_vec[i + 1] - 1].push_back({seq_id1, next_node_id1});

      size_type seq_id1 = h_seq_id[i + 1];
      node_type next_node_id1 = source[(*start_pos_map)[seq_id1] + position +
      1];
      sorted_seqs[h_keys_vec[i + 1] - 1].push_back({seq_id1, next_node_id1});

      size_type seq_id1 = h_seq_id[i + 1];
      node_type next_node_id1 = source[(*start_pos_map)[seq_id1] + position +
      1];
      sorted_seqs[h_keys_vec[i + 1] - 1].push_back({seq_id1, next_node_id1});

      size_type seq_id1 = h_seq_id[i + 1];
      node_type next_node_id1 = source[(*start_pos_map)[seq_id1] + position +
      1];
      sorted_seqs[h_keys_vec[i + 1] - 1].push_back({seq_id1, next_node_id1});
  }
  */
    /*
    size_type start = (seqs_left / unroll_num) * unroll_num;
    for (size_type i = end_counter + start; i < seqs_left + end_counter; ++i) {
      size_type seq_id = h_seq_id[i];
      node_type next_node_id = source[(*start_pos_map)[seq_id] + position + 1];
      sorted_seqs[h_keys_vec[i] - 1].push_back({seq_id, next_node_id});
    }
    */
    end = std::chrono::steady_clock::now();
    place_time +=
        std::chrono::duration_cast<std::chrono::microseconds>(end - begin)
            .count();
  }
  hipFree(d_source);
  thrust::device_free(d_keys);
  std::cout << "init_time: " << init_time << " [μs]\n";
  std::cout << "key_time: " << key_time << " [μs]\n";
  std::cout << "d2h_copy_time: " << d2h_copy_time << " [μs]\n";
  std::cout << "sort_time: " << sort_time << " [μs]\n";
  std::cout << "remove_time: " << remove_time << " [μs]\n";
  std::cout << "place_time: " << place_time << " [μs]\n";
  /* print sorted_seqs
  int i = 1;
  for (auto &vec : sorted_seqs) {
    std::cout << "node_id: " << i++ << " ";
    for (auto &item : vec) {
      std::cout << "(" << item.first << ", " << item.second << ") ";
    }
    std::cout << "\n";
  }
  */
  /* print a specific sequence
  node_type node_id = 2;
  auto node2_vec = sorted_seqs[node_id - 1];
  for (auto &item : node2_vec) {
    std::cout << "(" << item.first << ", " << item.second << ")\n";
  }
  */
  return sorted_seqs;
}

/*
// test section: get_int
node_type *d_test;
int start_idx = 99996850;
size_type vec_size = 10;
hipMalloc(&d_test, sizeof(node_type) * vec_size);
int test_block = (vec_size + thread_per_block - 1) / thread_per_block;
test_get_int<<<test_block, thread_per_block>>>(d_source, d_test, vec_size,
                                             start_idx, t_width);
for (int i = start_idx; i < start_idx + vec_size; ++i) {
std::cout << source[i] << " ";
}
// std::cout << source.data()[start_idx] << "\n";
std::cout << "\n========================\n";
node_type *d_test_H = new node_type[vec_size + 1];
hipMemcpy(d_test_H, d_test, sizeof(node_type) * vec_size,
         hipMemcpyDeviceToHost);
for (int i = 0; i < vec_size; ++i) {
std::cout << d_test_H[i] << " ";
}
hipFree(d_test);
delete[] d_test_H;
std::cout << "\n";
// test section: get_int
*/

/* test a specific sequence
for (size_type i = end_counter; i < (seqs_left / unroll_num);
     i += unroll_num) {
  size_type seq_id = h_seq_id[i];
  node_type next_node_id = source[(*start_pos_map)[seq_id] + position + 1];
  sorted_seqs[h_keys_vec[i] - 1].push_back({seq_id, next_node_id});
  if (seq_id == 716) {
    std::cout << "key: " << h_keys_vec[i] << "\n";
  }
}
*/
