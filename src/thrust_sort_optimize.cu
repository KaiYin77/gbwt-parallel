#include "hip/hip_runtime.h"
#include <chrono>
#include <thrust/device_free.h>
#include <thrust/device_malloc.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust_sort.cuh>

void print_vec(const thrust::host_vector<size_type> &vec) {
  for (auto &item : vec) {
    std::cout << item << " ";
  }
  std::cout << "\n";
}

__device__ uint64_t get_int(const uint64_t *m_data, const size_type idx,
                            const uint8_t len) {
  const uint64_t *word = m_data + (idx >> 6);
  const uint8_t offset = idx & 0x3F;
  uint64_t w1 = (*word) >> offset;
  if ((offset + len) > 64) { // if offset+len > 64
                             // w1 or w2 adepted:
    return w1 | ((*(word + 1) &
                  (1 << ((offset + len) & 0x3F)) - 1) // set higher bits zero
                 << (64 - offset));                   // move bits to the left
  } else {
    return w1 & ((1 << len) - 1);
  }
}

__global__ void test_get_int(const uint64_t *d_source, node_type *d_test,
                             const size_type vec_size,
                             const size_type start_idx, const int t_width,
                             const int thread_per_block) {
  size_type idx = blockIdx.x * thread_per_block + threadIdx.x;
  if (idx < vec_size) {
    d_test[idx] =
        get_int(d_source, size_type((start_idx + idx)) * t_width, t_width);
  }
  return;
}

__global__ void assign_key(const uint64_t *d_source,
                           const size_type *d_start_pos, node_type *d_keys,
                           size_type *d_seq_id, const size_type position,
                           const size_type seqs_size, const int t_width,
                           const int thread_per_block) {
  size_type idx = blockIdx.x * thread_per_block + threadIdx.x;
  if (idx < seqs_size) {
    d_keys[idx] = get_int(
        d_source, (d_start_pos[d_seq_id[idx]] + position) * t_width, t_width);
  }
  return;
}

struct is_zero {
  __host__ __device__ bool operator()(int x) { return (x == 0); }
};

std::vector<std::vector<std::pair<size_type, node_type>>>
radix_sort(const text_type &source, std::vector<size_type> &sequence_id,
           const std::unique_ptr<std::unordered_map<size_type, size_type>>
               &start_pos_map,
           const std::uint64_t total_nodes) {
  double init_time = 0, key_time = 0, h2d_copy_time = 0, sort_time = 0,
         d2h_copy_time = 0, remove_time = 0, place_time = 0;
  std::chrono::steady_clock::time_point begin, end;
  begin = std::chrono::steady_clock::now();

  std::vector<std::vector<std::pair<size_type, node_type>>> sorted_seqs;
  sorted_seqs.reserve(total_nodes);

  // ---- Prepare values to be used ---- //
  // width of the integers which are accessed via the [] operator
  const uint8_t t_width = source.width();
  const size_type source_size_byte = source.bit_size() / 8 + 1;
  const size_type seqs_size = (*start_pos_map).size(); // total sequence size

  // copy source to device memory
  uint64_t *d_source;
  hipMalloc(&d_source, source_size_byte);
  hipMemcpyAsync(d_source, source.data(), source_size_byte,
                  hipMemcpyHostToDevice);

  // copy start_position
  thrust::host_vector<size_type> start_pos;
  start_pos.reserve(seqs_size);
  for (auto &seq_id : sequence_id) {
    start_pos.push_back((*start_pos_map)[seq_id]);
  }
  thrust::device_vector<size_type> start_pos_vec = start_pos;
  size_type *d_start_pos = thrust::raw_pointer_cast(&start_pos_vec[0]);
  // thrust::device_ptr<size_type> d_start_pos = start_pos_vec.data();

  // copy sequence_id
  thrust::host_vector<size_type> h_seq_id(sequence_id);
  thrust::device_vector<size_type> seq_id_vec = h_seq_id;
  thrust::device_ptr<size_type> d_seq_id =
      thrust::device_pointer_cast(&seq_id_vec[0]);
  size_type *d_seq_id_raw = thrust::raw_pointer_cast(&seq_id_vec[0]);
  // thrust::device_ptr<size_type> d_seq_id = seq_id_vec.data();

  // use gpu to assign keys
  // there about 2000~ sequences passed in
  thrust::host_vector<node_type> h_keys_vec;
  h_keys_vec.reserve(seqs_size);
  thrust::device_ptr<node_type> d_keys =
      thrust::device_malloc(sizeof(node_type) * seqs_size);
  node_type *d_keys_raw = thrust::raw_pointer_cast(d_keys);
  // node_type *d_keys;
  // hipMalloc(&d_keys, sizeof(node_type) * seqs_size);

  size_type arr_start_idx = 0; // first index which is not an ENDMARKER
  size_type seqs_left =
      seqs_size; // sequences that have not reached the ENDMARKER
  const int thread_per_block = 256;
  hipDeviceSynchronize();
  end = std::chrono::steady_clock::now();
  init_time = std::chrono::duration_cast<std::chrono::microseconds>(end - begin)
                  .count();
  for (size_type position = 0; seqs_left > 0; ++position) {
    //---- Assign Keys ----//
    begin = std::chrono::steady_clock::now();
    const int block_per_grid =
        (seqs_left + thread_per_block - 1) / thread_per_block;
    assign_key<<<block_per_grid, thread_per_block>>>(
        d_source, d_start_pos, d_keys_raw + arr_start_idx,
        d_seq_id_raw + arr_start_idx, position, seqs_left, t_width,
        thread_per_block);
    end = std::chrono::steady_clock::now();
    key_time +=
        std::chrono::duration_cast<std::chrono::microseconds>(end - begin)
            .count();

    //---- Radix Sort ----//
    begin = std::chrono::steady_clock::now();
    thrust::stable_sort_by_key(d_keys + arr_start_idx, d_keys + seqs_size,
                               d_seq_id);
    end = std::chrono::steady_clock::now();
    sort_time +=
        std::chrono::duration_cast<std::chrono::microseconds>(end - begin)
            .count();

    //---- Remove paths that reaches the ENDMARKER(zero): version1 ----//
    begin = std::chrono::steady_clock::now();
    arr_start_idx = thrust::find_if_not(thrust::device, d_keys + arr_start_idx,
                                        d_keys + seqs_size, is_zero()) -
                    d_keys;
    seqs_left = seqs_size - arr_start_idx;
    end = std::chrono::steady_clock::now();
    remove_time +=
        std::chrono::duration_cast<std::chrono::microseconds>(end - begin)
            .count();
    if (seqs_left <= 0)
      break;

    //---- Copy keys and sequence id back to host ----//
    begin = std::chrono::steady_clock::now();
    thrust::copy(d_seq_id + arr_start_idx, d_seq_id + seqs_size,
                 h_seq_id.begin());
    thrust::copy(d_keys + arr_start_idx, d_keys + seqs_size,
                 h_keys_vec.begin());
    end = std::chrono::steady_clock::now();
    d2h_copy_time +=
        std::chrono::duration_cast<std::chrono::microseconds>(end - begin)
            .count();

    begin = std::chrono::steady_clock::now();
    for (size_type i = 0; i < seqs_left; ++i) {
      size_type seq_id = h_seq_id[i];
      node_type next_node_id = source[(*start_pos_map)[seq_id] + position + 1];
      sorted_seqs[h_keys_vec[i] - 1].push_back({seq_id, next_node_id});
    }
    end = std::chrono::steady_clock::now();
    place_time +=
        std::chrono::duration_cast<std::chrono::microseconds>(end - begin)
            .count();
  }
  hipFree(d_source);
  thrust::device_free(d_keys);
  std::cout << "init_time: " << init_time << " [μs]\n";
  std::cout << "key_time: " << key_time << " [μs]\n";
  std::cout << "d2h_copy_time: " << d2h_copy_time << " [μs]\n";
  std::cout << "sort_time: " << sort_time << " [μs]\n";
  std::cout << "remove_time: " << remove_time << " [μs]\n";
  std::cout << "place_time: " << place_time << " [μs]\n";
  /* print sorted_seqs
  int i = 1;
  for (auto &vec : sorted_seqs) {
    std::cout << "node_id: " << i++ << " ";
    for (auto &item : vec) {
      std::cout << "(" << item.first << ", " << item.second << ") ";
    }
    std::cout << "\n";
  }
  */
  return sorted_seqs;
}

/*
// test section: get_int
node_type *d_test;
int start_idx = 99996850;
size_type vec_size = 10;
hipMalloc(&d_test, sizeof(node_type) * vec_size);
int test_block = (vec_size + thread_per_block - 1) / thread_per_block;
test_get_int<<<test_block, thread_per_block>>>(d_source, d_test, vec_size,
                                               start_idx, t_width,
                                               thread_per_block);
for (int i = start_idx; i < start_idx + vec_size; ++i) {
  std::cout << source[i] << " ";
}
// std::cout << source.data()[start_idx] << "\n";
std::cout << "\n========================\n";
node_type *d_test_H = new node_type[vec_size + 1];
hipMemcpy(d_test_H, d_test, sizeof(node_type) * vec_size,
           hipMemcpyDeviceToHost);
for (int i = 0; i < vec_size; ++i) {
  std::cout << d_test_H[i] << " ";
}
hipFree(d_test);
delete[] d_test_H;
std::cout << "\n";
// test section: get_int
*/
